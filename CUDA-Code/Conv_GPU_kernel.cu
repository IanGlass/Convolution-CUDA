#include "hip/hip_runtime.h"
/*
 * File: Conv_GPU_kernel.cpp
 * 
 * Author: Ian Glass
 * 
 * Date: 14/10/2013
 * 
 * Course: ENCE 463
 * 
 * Description: Module for performing Autocorrelation on the GPU using 
 * convolution.
 * 
 * Usage: The module takes input data, width and height and returns 
 * the output matrix with the time taken.
 * 
 */

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h>
#include <helper_functions.h> // helper functions for SDK examples

typedef unsigned int  uint;
typedef unsigned char uchar;

//Defines the maximum number of threads per block
#define thread_limit 512

////////////////////////////////////////////////////////////////////////////////
// Complex operations
////////////////////////////////////////////////////////////////////////////////

//Complex addition on device
static __device__ __host__ inline hipDoubleComplex ComplexAdd(hipDoubleComplex a, hipDoubleComplex b) {
    hipDoubleComplex c;
    c.x = a.x + b.x;
    c.y = a.y + b.y;
    return c;
}

// Complex multiplication on device
static __device__ __host__ inline hipDoubleComplex ComplexMul(hipDoubleComplex a, hipDoubleComplex b) {
    hipDoubleComplex c;
    c.x = a.x * b.x - a.y * b.y;
    c.y = a.x * b.y + a.y * b.x;
    return c;
}

// Computes convolution on the device
__global__ void Convolve(hipDoubleComplex *signal, hipDoubleComplex *filter_kernel, int size, hipDoubleComplex *filtered_signal) {
    int minRadius = size / 2;
    int maxRadius = size - minRadius;
	
	//Find current position in matrix as 1D index
	int ID = blockIdx.x * blockDim.x + threadIdx.x;
	
    // Loop over output element indices
    filtered_signal[ID].x = filtered_signal[ID].y = 0;

    // Loop over convolution indices
    for (int j = - maxRadius + 1; j <= minRadius; ++j) {
		int k = ID + j;

        if (k >= 0 && k < size) {
			filtered_signal[ID] = ComplexAdd(filtered_signal[ID], ComplexMul(signal[k], filter_kernel[minRadius - j]));
        }
    }
}

//Zero-pads the input matrix and shifts it by (width-1)/2
__global__ void Pad(hipDoubleComplex *input, hipDoubleComplex *output, int width, int width_out, int height) {
	
	int j = (blockIdx.x * blockDim.x + threadIdx.x)%width_out;
	int i = (blockIdx.x * blockDim.x + threadIdx.x)/width_out;
	
	output[blockIdx.x * blockDim.x + threadIdx.x].x = 0;
	output[blockIdx.x * blockDim.x + threadIdx.x].y = 0;
	
	//fill matrix with input data for even size
	if ((i<height)&&(j<width)) {
		output[(j+((width-1)/2))+(i+((height-1)/2))*width_out].x = input[j+i*width].x;
		output[(j+((width-1)/2))+(i+((height-1)/2))*width_out].y = input[j+i*width].y;
	}
}

////////////////////////////////////////////////////////////////////////////////
// Module main
////////////////////////////////////////////////////////////////////////////////
extern "C" hipDoubleComplex *Conv_GPU_fn(int argc, char **argv, hipDoubleComplex *data, int width, int height, float *time) {
	
	int width_out = width*2-1;
	int height_out = height*2-1;

    printf("\n%s Starting...", argv[0]);

    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    int devID = findCudaDevice(argc, (const char **)argv);
    
    StopWatchInterface *timer = 0;
    sdkCreateTimer(&timer);
    sdkStartTimer(&timer);
    
    int size_padded = width_out*height_out;
    
    //Set thread conditions
    int num_threads = thread_limit;
    if (size_padded < thread_limit) {
		num_threads = size_padded;
	}
	//
	
    unsigned int mem_size = sizeof(hipDoubleComplex)*size_padded;
    
    //setup execution parameters
    //calculate required grid dimensions
    int grid_size = size_padded/thread_limit+1;
    dim3 grid(grid_size, 1, 1);
    dim3 threads(num_threads, 1, 1);      
    //
    
    //Move input data to device
    hipDoubleComplex *data_in;
    checkCudaErrors(hipMalloc((void **)&data_in, sizeof(hipDoubleComplex)*width*height));
    checkCudaErrors(hipMemcpy(data_in, data, sizeof(hipDoubleComplex)*width*height, hipMemcpyHostToDevice));
    //
    
    //Create zero padded matrix
    hipDoubleComplex *data_padded;
    checkCudaErrors(hipMalloc((void **)&data_padded, mem_size));
    Pad<<<grid,threads>>>(data_in,data_padded, width, width_out, height);
    //
		
	//Perform Convolution
    hipDoubleComplex *h_convolved_signal;
    checkCudaErrors(hipMalloc((void **)&h_convolved_signal, mem_size));
	Convolve<<<grid,threads>>>(data_padded, data_padded, size_padded, h_convolved_signal);
	//
	
	//Move result back to host
	hipDoubleComplex *conv_out = (hipDoubleComplex*) malloc(mem_size);
	checkCudaErrors(hipMemcpy(conv_out, h_convolved_signal, mem_size, hipMemcpyDeviceToHost));
	//	
	                 
    sdkStopTimer(&timer);
    *time = sdkGetTimerValue(&timer);
    sdkDeleteTimer(&timer);
    
    //Tidy up
    hipFree(data_in);
    hipFree(data_padded);
    hipFree(h_convolved_signal);        
    
    hipDeviceReset();
    
    return(conv_out);
}



